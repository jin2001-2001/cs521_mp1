#include "hip/hip_runtime.h"
#include "../include/utils.h"
#include <hip/hip_runtime.h>

#define NUM_RUNS 10

#define CUDA_CHECK(func)                                                     	   \
	do {                                                                           \
		hipError_t status = (func);                                               \
		if (status != hipSuccess) {                                               \
			printf("CUDA API failed at line %d with error: %s (%d)\n", __LINE__,   \
				hipGetErrorString(status), status);                               \
			exit(EXIT_FAILURE);                                                    \
		}                                                                          \
	} while (0)

#define CHECK(name) \
	float *d_Aref_ ## name, *d_Bref_ ## name, *d_Cref_ ## name; \
	std::cerr << "checking " << #name << std::endl; \
	CUDA_CHECK(hipMalloc(&d_Aref_ ## name, Ref::M * Ref::K * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_Bref_ ## name, Ref::K * Ref::N * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_Cref_ ## name, Ref::M * Ref::N * sizeof(float))); \
	CUDA_CHECK(hipMemcpy(d_Aref_ ## name, ref.A, Ref::M * Ref::K * sizeof(float), hipMemcpyHostToDevice)); \
	CUDA_CHECK(hipMemcpy(d_Bref_ ## name, ref.B, Ref::K * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
	float* d_Cref_INI_ ## name = new float[M * N](); \
	for (int i = 0; i < Ref::M; i++) { \
		for (int j = 0; j < Ref::N; j++) { \
			d_Cref_INI_ ## name[i * Ref::N + j] = 0; \
		} \
	} \
	CUDA_CHECK(hipMemcpy(d_Cref_ ## name, d_Cref_INI_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
	name(d_Aref_ ## name, d_Bref_ ## name, d_Cref_ ## name, Ref::M, Ref::N, Ref::K); \
	hipError_t err_c_ ## name = hipGetLastError(); \
	if (err_c_ ## name != hipSuccess) { \
		std::cerr << "CUDA Error: " << hipGetErrorString(err_c_ ## name) << std::endl; \
	} \
	CUDA_CHECK(hipMemcpy(refC, d_Cref_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyDeviceToHost)); \
	if (!ref.checkRef(refC)){ \
		std::cerr << "check ref failed!" << std::endl; \
	};

#define TIME(name) \
	float *d_A_ ## name, *d_B_ ## name, *d_C_ ## name; \
	CUDA_CHECK(hipMalloc(&d_A_ ## name, M * K * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_B_ ## name, K * N * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_C_ ## name, M * N * sizeof(float))); \
	CUDA_CHECK(hipMemcpy(d_A_ ## name, A, M * K * sizeof(float), hipMemcpyHostToDevice)); \
	CUDA_CHECK(hipMemcpy(d_B_ ## name, B, K * N * sizeof(float), hipMemcpyHostToDevice)); \
	hipEvent_t start_ ## name, end_ ## name; \
	hipEventCreate(&start_ ## name); \
	hipEventCreate(&end_ ## name); \
	float* d_C_INI_ ## name = new float[M * N](); \
	for (int i = 0; i < M; i++) { \
		for (int j = 0; j < N; j++) { \
			d_C_INI_ ## name[i * N + j] = 0; \
		} \
	} \
	for (int i = 0; i < 2; i++) \
	{ \
		CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, M * N * sizeof(float), hipMemcpyHostToDevice)); \
		name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
	} \
	hipError_t err_t_ ## name = hipGetLastError(); \
	if (err_t_ ## name != hipSuccess) { \
		std::cerr << "CUDA Error: " << hipGetErrorString(err_t_ ## name) << std::endl; \
	} \
	float milliseconds_ ## name = 0; \
	for (int i = 0; i < NUM_RUNS; i++) \
	{ \
		CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, M * N * sizeof(float), hipMemcpyHostToDevice)); \
		hipDeviceSynchronize(); \
		hipEventRecord(start_ ## name); \
		name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
		hipEventRecord(end_ ## name); \
		hipEventSynchronize(end_ ## name); \
		float milliseconds_ ## i = 0; \
		hipEventElapsedTime(&milliseconds_ ## i, start_ ## name, end_ ## name); \
		milliseconds_ ## name += milliseconds_ ## i; \
	} \
	hipMemcpy(C, d_C_ ## name, M * N * sizeof(float), hipMemcpyDeviceToHost); \
	std::cout << "Time taken for GEMM (GPU, " << #name <<"): " << milliseconds_ ## name / (float)NUM_RUNS << "ms" << std::endl; \
	hipFree(d_A_ ## name); \
	hipFree(d_B_ ## name); \
	hipFree(d_C_ ## name);

__global__ void gemm_gpu_o0_kernel(float* A, float* B, float *C, int M, int N, int K) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		for (int i = 0; i < M; i++) {
			for (int j = 0; j < N; j++) {
				for (int k = 0; k < K; k++) {
					C[i * N + j]  += A[i * K + k]  * B[k * N + j];
				}
			}
		}
    }
}

void gemm_gpu_o0(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 blockSize(1);
	dim3 gridSize(1);
	gemm_gpu_o0_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

// The scafolding for optimized GEMM implementations
__global__ void gemm_gpu_o1_kernel(float* A, float* B, float *C, int M, int N, int K) {
	// For C, the Row is M, Col is N, for A, row is M, col is K, For B Row is K, Col is N.
	int Col = blockIdx.x * blockDim.x + threadIdx.x;
	int Row = blockIdx.y * blockDim.y + threadIdx.y;

	if ((Row < M) && (Col < N)) {
		float Pvalue = 0;
 		for (int k = 0; k < K; ++k)
 			Pvalue += A[Row*K+k] * B[k*N+Col];
	  	C[Row * N + Col] = Pvalue;
	}

}

#define BlockW 16
void gemm_gpu_o1(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size

	dim3 dimGrid(ceil((1.0*M)/BlockW),ceil((1.0*N)/BlockW), 1);
	dim3 dimBlock(BlockW, BlockW, 1);
	gemm_gpu_o1_kernel<<<dimGrid, dimBlock>>>(A, B, C, M, N, K);

}

#define TILE_WIDTH2 16
__global__ void gemm_gpu_o2_kernel(float* A, float* B, float *C, int M, int N, int K) {

	__shared__ float subTileA[TILE_WIDTH2][TILE_WIDTH2];
	__shared__ float subTileB[TILE_WIDTH2][TILE_WIDTH2];
	int bx = blockIdx.x;  int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	float Pvalue = 0;

	//big loop iterate tile by tile
	for (int q = 0; q < (ceil((float)K/TILE_WIDTH2)); ++q) {
       // Collaborative loading of M and N tiles into shared memory
		if (Row < M && (q*TILE_WIDTH2 + tx) < K)
			subTileA[ty][tx] = M[Row*K + (q*TILE_WIDTH2+tx)];
		else
			subTileA[ty][tx] = 0;

		if (Col < N && (q*TILE_WIDTH2 + ty) < K)
			subTileB[ty][tx] = N[(q*TILE_WIDTH2+ty)*N+Col];
		else
			subTileB[ty][tx] = 0;
		__syncthreads();  //must needed for correct tile loading instead of overload
		for (int k = 0; k < TILE_WIDTH2; ++k)
			Pvalue += subTileM[ty][k] * subTileN[k][tx];
		__syncthreads();  //make sure everyone complete calculate
	}
	if(Row < M && Col < N)
		C[Row*N+Col] = Pvalue;

}

void gemm_gpu_o2(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 dimGrid(ceil((1.0*M)/BlockW),ceil((1.0*N)/BlockW), 1);
	dim3 dimBlock(BlockW, BlockW, 1);
	gemm_gpu_o2_kernel<<<dimGrid, dimBlock>>>(A, B, C, M, N, K);
}


#define BlockW3 32
#define TILE_WIDTH3 32

__global__ void gemm_gpu_o3_kernel(float* A, float* B, float *C, int M, int N, int K) {
	//code is the same as o2 kernel, but with hyperparameter change...
	__shared__ float subTileA[TILE_WIDTH3][TILE_WIDTH3];
	__shared__ float subTileB[TILE_WIDTH3][TILE_WIDTH3];
	int bx = blockIdx.x;  int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	float Pvalue = 0;

	//big loop iterate tile by tile
	for (int q = 0; q < (ceil((float)K/TILE_WIDTH3)); ++q) {
       // Collaborative loading of M and N tiles into shared memory
		if (Row < M && (q*TILE_WIDTH3 + tx) < K)
			subTileA[ty][tx] = M[Row*K + (q*TILE_WIDTH3+tx)];
		else
			subTileA[ty][tx] = 0;

		if (Col < N && (q*TILE_WIDTH3 + ty) < K)
			subTileB[ty][tx] = N[(q*TILE_WIDTH3+ty)*N+Col];
		else
			subTileB[ty][tx] = 0;
		__syncthreads();  //must needed for correct tile loading instead of overload
		for (int k = 0; k < TILE_WIDTH3; ++k)
			Pvalue += subTileM[ty][k] * subTileN[k][tx];
		__syncthreads();  //make sure everyone complete calculate
	}
	if(Row < M && Col < N)
		C[Row*N+Col] = Pvalue;

}
void gemm_gpu_o3(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 dimGrid(ceil((1.0*M)/BlockW3),ceil((1.0*N)/BlockW3), 1);
	dim3 dimBlock(BlockW3, BlockW3, 1);
	gemm_gpu_o3_kernel<<<dimGrid, dimBlock>>>(A, B, C, M, N, K);
}



int main(int argc, char* argv[]) {
	if (argc < 3) {
		std::cout << "Usage: mp1 <M> <N> <K>" << std::endl;
		return 1;
	}

	int M = atoi(argv[1]);
	int N = atoi(argv[2]);
	int K = atoi(argv[3]);

	// int runs = atoi(argv[3]);
	float* A = new float[M * K]();
	float* B = new float[K * N]();
	float* C = new float[M * N]();

	fillRandom(A, M * K);
	fillRandom(B, K * N);

	/// GPU Implementation
        // Check if implementation is correct
	auto ref = Ref();
	float* refC = new float[Ref::M * Ref::N]();
 	CHECK(gemm_gpu_o0)
	CHECK(gemm_gpu_o1)
	CHECK(gemm_gpu_o2)
	CHECK(gemm_gpu_o3)

	// Actual run
 	TIME(gemm_gpu_o0)
	TIME(gemm_gpu_o1)
	TIME(gemm_gpu_o2)
	TIME(gemm_gpu_o3)

	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);

	delete[] A;
	delete[] B;
	delete[] C;

	return 0;
}